#include "hip/hip_runtime.h"
#include "HelloWorld.cuh"


// function is executed parallel in blocks
// or __device__
__global__ void copy_string_kernel(char * dp_dst, char * dp_src, size_t const size)
{
    if ((dp_dst != nullptr) && (dp_src != nullptr))
    {
        auto const i = blockIdx.x * blockDim.x + threadIdx.x;

        if (i < size) {
            dp_dst[i] = dp_src[i];
        }
    }
}

__host__ void copy_string_kernel_wrapper(size_t const threads_in_block,
    size_t const blocks_in_grid,
    char * dp_dst,
    char * dp_src,
    size_t const size)
{
    copy_string_kernel <<<threads_in_block, blocks_in_grid>>> (dp_dst, dp_src, size);
}