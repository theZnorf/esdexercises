#include "hip/hip_runtime.h"
//       $Id: gaussian_filter.cu 30314 2016-01-03 15:42:34Z p20068 $
//      $URL: https://svn01.fh-hagenberg.at/bin/cepheiden/vocational/teaching/ESD/SPS3/2015-WS/Ablauf/src/Filters/src/device/gaussian_filter.cu $
// $Revision: 30314 $
//     $Date: 2016-01-03 16:42:34 +0100 (So., 03 Jän 2016) $
//   Creator: Ch. Angerer, NVIDIA, J. Demouth, NVIDIA, peter.kulczycki<AT>fh-hagenberg.at
//   $Author: p20068 $

#include "./gaussian_filter.h"

// -------------------------------------------------------------------------------------------------

__constant__ int const filter_v1 [7][7] = {
   {1, 2,  3,  4,  3, 2, 1},
   {2, 4,  6,  8,  6, 4, 2},
   {3, 6,  9, 12,  9, 6, 3},
   {4, 8, 12, 16, 12, 8, 4},
   {3, 6,  9, 12,  9, 6, 3},
   {2, 4,  6,  8,  6, 4, 2},
   {1, 2,  3,  4,  3, 2, 1}
};

__global__ void gaussian_filter_kernel_v1 (int const w, int const h, pfc::byte_t * const dp_dst, pfc::byte_t const * const dp_src) {

// position of the thread in the image

   int const x = pfc::global_thread_idx_x ();
   int const y = pfc::global_thread_idx_y ();

   if (pfc::inside_rect (x, y, w, h)) {   // early exit if the thread is not in the image

//    load my 48 neighbors and myself

      int n [7][7];   // = {}   // initialization costs time

      for (int j = -3; j <= 3; ++j) {
         for (int i = -3; i <= 3; ++i) {
            n[j + 3][i + 3] = pfc::inside_rect (x + i, y + j, w, h) ? dp_src[(y + j) * w + (x + i)] : 0;
         }
      }

//    compute the convolution

      int p = 0;

      for (int j = 0; j < 7; ++j) {
         for (int i = 0; i < 7; ++i) {
            p += filter_v1[j][i] * n[j][i];
         }
      }

//    store the result

      dp_dst[y * w + x] = p / 256;
   }
}

// -------------------------------------------------------------------------------------------------

#undef LBOUNDS

#if   defined CUDA_ARCH_300
   #define LBOUNDS __launch_bounds__ (128, 10)   /* max. threads per block, min. blocks per multiprocessor */

#elif defined CUDA_ARCH_200
   #define LBOUNDS __launch_bounds__ (128,  8)   /* max. threads per block, min. blocks per multiprocessor */

#else
   #define LBOUNDS
#endif

LBOUNDS __global__ void gaussian_filter_kernel_v2 (int const w, int const h, pfc::byte_t * const dp_dst, pfc::byte_t const * const dp_src) {

// position of the thread in the image

   int const x = pfc::global_thread_idx_x ();
   int const y = pfc::global_thread_idx_y ();

   if (pfc::inside_rect (x, y, w, h)) {   // early exit if the thread is not in the image

//    load my 48 neighbors and myself

      int n [7][7];   // = {}   // initialization costs time

      for (int j = -3; j <= 3; ++j) {
         for (int i = -3; i <= 3; ++i) {
            n[j + 3][i + 3] = pfc::inside_rect (x + i, y + j, w, h) ? dp_src[(y + j) * w + (x + i)] : 0;
         }
      }

//    compute the convolution

      int p = 0;

      for (int j = 0; j < 7; ++j) {
         for (int i = 0; i < 7; ++i) {
            p += filter_v1[j][i] * n[j][i];
         }
      }

//    store the result

      dp_dst[y * w + x] = p / 256;
   }
}

// -------------------------------------------------------------------------------------------------

__global__ void gaussian_filter_kernel_v3 (int const w, int const h, pfc::byte_t * const dp_dst, pfc::byte_t const * const dp_src) {
   __shared__ pfc::byte_t adj [10][64];   // allocate static shared memory

// position of the thread in the image

   int const x = pfc::global_thread_idx_x ();
   int const y = pfc::global_thread_idx_y ();

// load adjacent pixels into shared memory

   pfc::byte_t * p_pel = &adj[threadIdx.y][threadIdx.x];

   for (int i = y - 3; i <= blockIdx.y * blockDim.y + 6; i += 4, p_pel += 4 * 64) {
      p_pel[ 0] = pfc::inside_rect (x      - 3, i, w, h) ? dp_src[i * w + (x      - 3)] : 0;
      p_pel[32] = pfc::inside_rect (x + 32 - 3, i, w, h) ? dp_src[i * w + (x + 32 - 3)] : 0;
   }

   __syncthreads ();   // wait for all threads to finish loading

// load my 48 neighbors and myself

   int n [7][7];   // = {}   // initialization costs time

   for (int j = 0; j < 7; ++j) {
      for (int i = 0; i < 7; ++i) {
         n[j][i] = adj[threadIdx.y + j][threadIdx.x + i];
      }
   }

// compute the convolution

   int p = 0;

   for (int j = 0; j < 7; ++j) {
      for (int i = 0; i < 7; ++i) {
         p += filter_v1[j][i] * n[j][i];
      }
   }

// store the result

   if (pfc::inside_rect (x, y, w, h)) dp_dst[y * w + x] = p / 256;
}

// -------------------------------------------------------------------------------------------------

__global__ void gaussian_filter_kernel_v4 (int const w, int const h, pfc::byte_t * const dp_dst, pfc::byte_t const * __restrict__ const dp_src) {
   __shared__ float adj [32][40];   // allocate static shared memory

// position of the thread in the image

   int const x = 1 * (blockIdx.x * blockDim.x) + threadIdx.x;
   int const y = 2 * (blockIdx.y * blockDim.y) + threadIdx.y;

// each thread loads 2 * 4 pixels into a temporary buffer

   int const read_x = blockIdx.x * blockDim.x + 2 * threadIdx.x - 4;   // index of pixel to load (-4 for alignment, should be -3?)

   uchar2 const p0 = pfc::inside_rect (read_x, y -  3, w, h) ? *reinterpret_cast <uchar2 const *> (dp_src + ((y -  3) * w + read_x)) : make_uchar2 (0, 0);
   uchar2 const p1 = pfc::inside_rect (read_x, y +  5, w, h) ? *reinterpret_cast <uchar2 const *> (dp_src + ((y +  5) * w + read_x)) : make_uchar2 (0, 0);
   uchar2 const p2 = pfc::inside_rect (read_x, y + 13, w, h) ? *reinterpret_cast <uchar2 const *> (dp_src + ((y + 13) * w + read_x)) : make_uchar2 (0, 0);
   uchar2 const p3 = pfc::inside_rect (read_x, y + 21, w, h) ? *reinterpret_cast <uchar2 const *> (dp_src + ((y + 21) * w + read_x)) : make_uchar2 (0, 0);

// only the first 20 threads of a block load the pixels from the buffers into shared memory

   if (threadIdx.x < 20) {
      reinterpret_cast <float2 *> (adj[threadIdx.y +  0])[threadIdx.x] = make_float2 (p0.x, p0.y);
      reinterpret_cast <float2 *> (adj[threadIdx.y +  8])[threadIdx.x] = make_float2 (p1.x, p1.y);
      reinterpret_cast <float2 *> (adj[threadIdx.y + 16])[threadIdx.x] = make_float2 (p2.x, p2.y);
      reinterpret_cast <float2 *> (adj[threadIdx.y + 24])[threadIdx.x] = make_float2 (p3.x, p3.y);
   }

   __syncthreads ();   // wait for all threads to finish loading

// load my 48 neighbors and myself

   float n [8][7];   // = {}   // initialization costs time

   for (int j = 0; j <= 7; ++j) {
      for (int i = 0; i < 7; ++i) {
         n[j][i] = adj[2 * threadIdx.y + j][threadIdx.x + i];
      }
   }

// compute the convolution

// float p [2] = {};
   float q0    = {};
   float q1    = {};

   for (int j = 0; j < 7; ++j) {
      for (int i = 0; i < 7; ++i) {
         q0 += filter_v1[j][i] * n[j + 0][i];
         q1 += filter_v1[j][i] * n[j + 1][i];
      }
   }

// store the results

   int const write_y = y + threadIdx.y;

   if (pfc::inside_rect (x, write_y + 0, w, h)) dp_dst[(write_y + 0) * w + x] = int (q0) >> 8;
   if (pfc::inside_rect (x, write_y + 1, w, h)) dp_dst[(write_y + 1) * w + x] = int (q1) >> 8;
}

// -------------------------------------------------------------------------------------------------

__global__ void gaussian_filter_kernel_v5 (int const w, int const h, pfc::byte_t * const dp_dst, pfc::byte_t const * __restrict__ const dp_src) {
   __shared__ int adj [16][38];   // allocate static shared memory

// position of the thread in the image

   int const x = pfc::global_thread_idx_x ();
   int const y = pfc::global_thread_idx_y ();

// each thread loads two pixels into a temporary buffer

   int const p0 = pfc::inside_rect (x - 3, y - 3, w, h) ? dp_src[(y - 3) * w + x - 3] : 0;
   int const p1 = pfc::inside_rect (x - 3, y + 5, w, h) ? dp_src[(y + 5) * w + x - 3] : 0;

// load extra pixels per row

   int const p2 = (pfc::inside_rect (x + 29, y - 3, w, h) && threadIdx.x < 6) ? dp_src[(y - 3) * w + x + 29] : 0;
   int const p3 = (pfc::inside_rect (x + 29, y + 5, w, h) && threadIdx.x < 6) ? dp_src[(y + 5) * w + x + 29] : 0;

// load adjacent pixels into shared memory

   adj[threadIdx.y + 0][threadIdx.x] = p0;
   adj[threadIdx.y + 8][threadIdx.x] = p1;

// load extra pixels into shared memory

   if (threadIdx.x < 6) {
      adj[threadIdx.y + 0][threadIdx.x + 32] = p2;
      adj[threadIdx.y + 8][threadIdx.x + 32] = p3;
   }

   __syncthreads ();   // wait for all threads to finish loading

// compute the horizontal convolution

   int n0 [7];   // = {}   // initialization costs time
   int n1 [7];   // = {}   // initialization costs time

   for (int i = 0; i < 7; ++i) {
      n0[i] = adj[threadIdx.y + 0][threadIdx.x + i];
      n1[i] = adj[threadIdx.y + 8][threadIdx.x + i];
   }

// int p [2] = {};
   int q0    = {};
   int q1    = {};

   for (int i = 0; i < 7; ++i) {
      q0 += filter_v1[0][i] * n0[i];
      q1 += filter_v1[0][i] * n1[i];
   }

   __syncthreads ();   // make sure the results are in shared memory

// write the results back to shared memory

   adj[threadIdx.y + 0][threadIdx.x] = q0;
   adj[threadIdx.y + 8][threadIdx.x] = q1;

   __syncthreads ();   // make sure the results are in shared memory

// compute the vertical convolution

   int n [7];   // = {}   // initialization costs time

   for (int i = 0; i < 7; ++i) {
      n[i] = adj[threadIdx.y + i][threadIdx.x];
   }

   int q = 0;

   for (int i = 0; i < 7; ++i) {
      q += filter_v1[i][0] * n[i];
   }

// store the results

   if (pfc::inside_rect (x, y, w, h)) dp_dst[y * w + x] = q >> 8;
}

// -------------------------------------------------------------------------------------------------

__global__ void gaussian_filter_kernel_v6 (int const w, int const h, pfc::byte_t * const dp_dst, pfc::byte_t const * __restrict__ const dp_src) {
   __shared__ int adj [32][40];   // allocate static shared memory

// position of the thread in the image

   int const x = 1 * (blockIdx.x * blockDim.x) + threadIdx.x;
   int const y = 2 * (blockIdx.y * blockDim.y) + threadIdx.y;

// each thread loads 2 * 4 pixels into a temporary buffer

   int const read_x = blockIdx.x * blockDim.x + 2 * threadIdx.x - 4;   // index of pixel to load (-4 for alignment, should be -3?)

   uchar2 const p0 = pfc::inside_rect (read_x, y -  3, w, h) ? *reinterpret_cast <uchar2 const *> (dp_src + ((y -  3) * w + read_x)) : make_uchar2 (0, 0);
   uchar2 const p1 = pfc::inside_rect (read_x, y +  5, w, h) ? *reinterpret_cast <uchar2 const *> (dp_src + ((y +  5) * w + read_x)) : make_uchar2 (0, 0);
   uchar2 const p2 = pfc::inside_rect (read_x, y + 13, w, h) ? *reinterpret_cast <uchar2 const *> (dp_src + ((y + 13) * w + read_x)) : make_uchar2 (0, 0);
   uchar2 const p3 = pfc::inside_rect (read_x, y + 21, w, h) ? *reinterpret_cast <uchar2 const *> (dp_src + ((y + 21) * w + read_x)) : make_uchar2 (0, 0);

// only the first 20 threads of a block load the pixels from the buffers into shared memory

   if (threadIdx.x < 20) {
      reinterpret_cast <int2 *> (adj[threadIdx.y +  0])[threadIdx.x] = make_int2 (p0.x, p0.y);
      reinterpret_cast <int2 *> (adj[threadIdx.y +  8])[threadIdx.x] = make_int2 (p1.x, p1.y);
      reinterpret_cast <int2 *> (adj[threadIdx.y + 16])[threadIdx.x] = make_int2 (p2.x, p2.y);
      reinterpret_cast <int2 *> (adj[threadIdx.y + 24])[threadIdx.x] = make_int2 (p3.x, p3.y);
   }

   __syncthreads ();   // wait for all threads to finish loading

// compute the horizontal convolution

   int n0 [7];   // = {}   // initialization costs time
   int n1 [7];   // = {}   // initialization costs time
   int n2 [7];   // = {}   // initialization costs time
   int n3 [7];   // = {}   // initialization costs time

   for (int i = 0; i < 7; ++i) {
      n0[i] = adj[threadIdx.y +  0][threadIdx.x + i + 1];   // +1 because of alignment constraints when loading pixels
      n1[i] = adj[threadIdx.y +  8][threadIdx.x + i + 1];   //
      n2[i] = adj[threadIdx.y + 16][threadIdx.x + i + 1];   //
      n3[i] = adj[threadIdx.y + 24][threadIdx.x + i + 1];   //
   }

   int p [4] = {};

   for (int i = 0; i < 7; ++i) {
      p[0] += filter_v1[0][i] * n0[i];
      p[1] += filter_v1[0][i] * n1[i];
      p[2] += filter_v1[0][i] * n2[i];
      p[3] += filter_v1[0][i] * n3[i];
   }

   __syncthreads ();   // make sure the results are in shared memory

// write the results back to shared memory

   adj[threadIdx.y +  0][threadIdx.x] = p[0];
   adj[threadIdx.y +  8][threadIdx.x] = p[1];
   adj[threadIdx.y + 16][threadIdx.x] = p[2];
   adj[threadIdx.y + 24][threadIdx.x] = p[3];

   __syncthreads ();   // make sure the results are in shared memory

// compute the vertical convolution

   int n [8];   // = {}   // initialization costs time

   for (int i = 0; i <= 7; ++i) {
      n[i] = adj[2 * threadIdx.y + i][threadIdx.x];
   }

// int q [2] = {};
   int q0    = {};
   int q1    = {};

   for (int i = 0; i < 7; ++i) {
      q0 += filter_v1[i][0] * n[i + 0];
      q1 += filter_v1[i][0] * n[i + 1];
   }

// store the results

   int const write_y = y + threadIdx.y;

   if (pfc::inside_rect (x, write_y + 0, w, h)) dp_dst[(write_y + 0) * w + x] = q0 >> 8;
   if (pfc::inside_rect (x, write_y + 1, w, h)) dp_dst[(write_y + 1) * w + x] = q1 >> 8;
}

// -------------------------------------------------------------------------------------------------

__constant__ float const filter_v2 [7] = {
   1, 2, 3, 4, 3, 2, 1
};

__global__ void gaussian_filter_kernel_v7 (int const w, int const h, pfc::byte_t * const dp_dst, pfc::byte_t const * __restrict__ const dp_src) {
   __shared__ float adj [32][40];   // allocate static shared memory

// position of the thread in the image

   int const x = 1 * (blockIdx.x * blockDim.x) + threadIdx.x;
   int const y = 2 * (blockIdx.y * blockDim.y) + threadIdx.y;

// each thread loads 2 * 4 pixels into a temporary buffer

   int const read_x = blockIdx.x * blockDim.x + 2 * threadIdx.x - 4;   // index of pixel to load (-4 for alignment, should be -3?)

   uchar2 const p0 = pfc::inside_rect (read_x, y -  3, w, h) ? *reinterpret_cast <uchar2 const *> (dp_src + ((y -  3) * w + read_x)) : make_uchar2 (0, 0);
   uchar2 const p1 = pfc::inside_rect (read_x, y +  5, w, h) ? *reinterpret_cast <uchar2 const *> (dp_src + ((y +  5) * w + read_x)) : make_uchar2 (0, 0);
   uchar2 const p2 = pfc::inside_rect (read_x, y + 13, w, h) ? *reinterpret_cast <uchar2 const *> (dp_src + ((y + 13) * w + read_x)) : make_uchar2 (0, 0);
   uchar2 const p3 = pfc::inside_rect (read_x, y + 21, w, h) ? *reinterpret_cast <uchar2 const *> (dp_src + ((y + 21) * w + read_x)) : make_uchar2 (0, 0);

// only the first 20 threads of a block load the pixels from the buffers into shared memory

   if (threadIdx.x < 20) {
      reinterpret_cast <float2 *> (adj[threadIdx.y +  0])[threadIdx.x] = make_float2 (p0.x, p0.y);
      reinterpret_cast <float2 *> (adj[threadIdx.y +  8])[threadIdx.x] = make_float2 (p1.x, p1.y);
      reinterpret_cast <float2 *> (adj[threadIdx.y + 16])[threadIdx.x] = make_float2 (p2.x, p2.y);
      reinterpret_cast <float2 *> (adj[threadIdx.y + 24])[threadIdx.x] = make_float2 (p3.x, p3.y);
   }

   __syncthreads ();   // wait for all threads to finish loading

// compute the horizontal convolution

   float n0 [7];   // = {}   // initialization costs time
   float n1 [7];   // = {}   // initialization costs time
   float n2 [7];   // = {}   // initialization costs time
   float n3 [7];   // = {}   // initialization costs time

   for (int i = 0; i < 7; ++i) {
      n0[i] = adj[threadIdx.y +  0][threadIdx.x + i + 1];   // +1 because of alignment constraints when loading pixels
      n1[i] = adj[threadIdx.y +  8][threadIdx.x + i + 1];   //
      n2[i] = adj[threadIdx.y + 16][threadIdx.x + i + 1];   //
      n3[i] = adj[threadIdx.y + 24][threadIdx.x + i + 1];   //
   }

   float p [4] = {};

   for (int i = 0; i < 7; ++i) {
      p[0] += filter_v2[i] * n0[i];
      p[1] += filter_v2[i] * n1[i];
      p[2] += filter_v2[i] * n2[i];
      p[3] += filter_v2[i] * n3[i];
   }

   __syncthreads ();   // make sure the results are in shared memory

// write the results back to shared memory

   adj[threadIdx.y +  0][threadIdx.x] = p[0];
   adj[threadIdx.y +  8][threadIdx.x] = p[1];
   adj[threadIdx.y + 16][threadIdx.x] = p[2];
   adj[threadIdx.y + 24][threadIdx.x] = p[3];

   __syncthreads ();   // make sure the results are in shared memory

// compute the vertical convolution

   float n [8];   // = {}   // initialization costs time

   for (int i = 0; i <= 7; ++i) {
      n[i] = adj[2 * threadIdx.y + i][threadIdx.x];
   }

// float q [2] = {};
   float q0    = {};
   float q1    = {};

   for (int i = 0; i < 7; ++i) {
      q0 += filter_v2[i] * n[i + 0];
      q1 += filter_v2[i] * n[i + 1];
   }

// store the results

   int const write_y = y + threadIdx.y;

   if (pfc::inside_rect (x, write_y + 0, w, h)) dp_dst[(write_y + 0) * w + x] = int (q0) >> 8;
   if (pfc::inside_rect (x, write_y + 1, w, h)) dp_dst[(write_y + 1) * w + x] = int (q1) >> 8;
}

// -------------------------------------------------------------------------------------------------

void gaussian_filter_kernel_v1_wrapper (pfc::byte_t * const dp_dst, pfc::byte_t const * const dp_src) {
   PFC_CUDA_CHECK (hipFuncSetCacheConfig(reinterpret_cast<const void*>(gaussian_filter_kernel_v1), hipFuncCachePreferL1));

   gaussian_filter_kernel_v1
      <<<pfc::grid_size (pfc::config::block_size_blur ()), pfc::config::block_size_blur ()>>>
      (PFC_CONFIG.img_w, PFC_CONFIG.img_h, dp_dst, dp_src);

   PFC_CUDA_CHECK (hipGetLastError ());
}

void gaussian_filter_kernel_v2_wrapper (pfc::byte_t * const dp_dst, pfc::byte_t const * const dp_src) {
   PFC_CUDA_CHECK (hipFuncSetCacheConfig(reinterpret_cast<const void*>(gaussian_filter_kernel_v2), hipFuncCachePreferL1));

   gaussian_filter_kernel_v2
      <<<pfc::grid_size (pfc::config::block_size_blur ()), pfc::config::block_size_blur ()>>>
      (PFC_CONFIG.img_w, PFC_CONFIG.img_h, dp_dst, dp_src);

   PFC_CUDA_CHECK (hipGetLastError ());
}

void gaussian_filter_kernel_v3_wrapper (pfc::byte_t * const dp_dst, pfc::byte_t const * const dp_src) {
   PFC_CUDA_CHECK (hipFuncSetCacheConfig(reinterpret_cast<const void*>(gaussian_filter_kernel_v3), hipFuncCachePreferShared));

   gaussian_filter_kernel_v3
      <<<pfc::grid_size (pfc::config::block_size_blur ()), pfc::config::block_size_blur ()>>>
      (PFC_CONFIG.img_w, PFC_CONFIG.img_h, dp_dst, dp_src);

   PFC_CUDA_CHECK (hipGetLastError ());
}

void gaussian_filter_kernel_v4_wrapper (pfc::byte_t * const dp_dst, pfc::byte_t const * __restrict__ const dp_src) {
   auto const tmp      = pfc::grid_size (pfc::config::block_size_blur ());
   auto const grid_dim = dim3 (tmp.x, tmp.y / 2);

   PFC_CUDA_CHECK (hipFuncSetCacheConfig(reinterpret_cast<const void*>(gaussian_filter_kernel_v4), hipFuncCachePreferShared));

   gaussian_filter_kernel_v4
      <<<grid_dim, pfc::config::block_size_blur ()>>>
      (PFC_CONFIG.img_w, PFC_CONFIG.img_h, dp_dst, dp_src);

   PFC_CUDA_CHECK (hipGetLastError ());
}

void gaussian_filter_kernel_v5_wrapper (pfc::byte_t * const dp_dst, pfc::byte_t const * __restrict__ const dp_src) {
   PFC_CUDA_CHECK (hipFuncSetCacheConfig(reinterpret_cast<const void*>(gaussian_filter_kernel_v5), hipFuncCachePreferShared));

   gaussian_filter_kernel_v5
      <<<pfc::grid_size (pfc::config::block_size_blur ()), pfc::config::block_size_blur ()>>>
      (PFC_CONFIG.img_w, PFC_CONFIG.img_h, dp_dst, dp_src);

   PFC_CUDA_CHECK (hipGetLastError ());
}

void gaussian_filter_kernel_v6_wrapper (pfc::byte_t * const dp_dst, pfc::byte_t const * __restrict__ const dp_src) {
   PFC_CUDA_CHECK (hipFuncSetCacheConfig(reinterpret_cast<const void*>(gaussian_filter_kernel_v6), hipFuncCachePreferShared));
   PFC_CUDA_CHECK (hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(gaussian_filter_kernel_v6), hipSharedMemBankSizeDefault));

   if (pfc::config::code_version ().is (9)) {
      PFC_CUDA_CHECK (hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(gaussian_filter_kernel_v6), hipSharedMemBankSizeEightByte));
   };

   auto const tmp      = pfc::grid_size (pfc::config::block_size_blur ());
   auto const grid_dim = dim3 (tmp.x, tmp.y / 2);

   gaussian_filter_kernel_v6
      <<<grid_dim, pfc::config::block_size_blur ()>>>
      (PFC_CONFIG.img_w, PFC_CONFIG.img_h, dp_dst, dp_src);

   PFC_CUDA_CHECK (hipGetLastError ());
}

void gaussian_filter_kernel_v7_wrapper (pfc::byte_t * const dp_dst, pfc::byte_t const * __restrict__ const dp_src) {
   PFC_CUDA_CHECK (hipFuncSetCacheConfig(reinterpret_cast<const void*>(gaussian_filter_kernel_v7), hipFuncCachePreferShared));
   PFC_CUDA_CHECK (hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(gaussian_filter_kernel_v7), hipSharedMemBankSizeEightByte));

   auto const tmp      = pfc::grid_size (pfc::config::block_size_blur ());
   auto const grid_dim = dim3 (tmp.x, tmp.y / 2);

   gaussian_filter_kernel_v7
      <<<grid_dim, pfc::config::block_size_blur ()>>>
      (PFC_CONFIG.img_w, PFC_CONFIG.img_h, dp_dst, dp_src);

   PFC_CUDA_CHECK (hipGetLastError ());

   PFC_CUDA_CHECK (hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(gaussian_filter_kernel_v7), hipSharedMemBankSizeDefault));
}
