#include "hip/hip_runtime.h"
//       $Id: sobel_filter.cu 30314 2016-01-03 15:42:34Z p20068 $
//      $URL: https://svn01.fh-hagenberg.at/bin/cepheiden/vocational/teaching/ESD/SPS3/2015-WS/Ablauf/src/Filters/src/device/sobel_filter.cu $
// $Revision: 30314 $
//     $Date: 2016-01-03 16:42:34 +0100 (So., 03 Jän 2016) $
//   Creator: Ch. Angerer, NVIDIA, J. Demouth, NVIDIA, peter.kulczycki<AT>fh-hagenberg.at
//   $Author: p20068 $

#include "./sobel_filter.h"

// -------------------------------------------------------------------------------------------------

namespace {
   __constant__ int const filter_v1_x [3][3] = {
      {-1,  0,  1},
      {-2,  0,  2},
      {-1,  0,  1}
   };

   __constant__ int const filter_v1_y [3][3] = {
      { 1,  2,  1},
      { 0,  0,  0},
      {-1, -2, -1}
   };
}

__global__ void sobel_filter_kernel_v1 (int const w, int const h, pfc::byte_t * const dp_dst, pfc::byte_t const * const dp_src) {

// position of the thread in the image

   int const x = pfc::global_thread_idx_x ();
   int const y = pfc::global_thread_idx_y ();

   if (pfc::inside_rect (x, y, w, h)) {   // early exit if the thread is not in the image

//    load my eight neighbors and myself

      int n [3][3];   // = {}   // initialization costs time

      for (int j = -1; j <= 1 ; ++j) {
         for (int i = -1; i <= 1; ++i) {
            n[j + 1][i + 1] = pfc::inside_rect (x + i, y + j, w, h) ? dp_src[(y + j) * w + (x + i)] : 0;
         }
      }

//    compute the convolution

      int gx = 0;
      int gy = 0;

      for (int j = 0; j < 3; ++j) {
         for (int i = 0; i < 3; ++i) {
            gx += filter_v1_x [j][i] * n[j][i];
            gy += filter_v1_y [j][i] * n[j][i];
         }
      }

//    compute the gradient and store the result

      dp_dst[y * w + x] = pfc::min (255.0f, sqrtf (gx * gx + gy * gy));
   }
}

// -------------------------------------------------------------------------------------------------

namespace {
   __constant__ float const filter_v2_x [3][3] = {
      {-1,  0,  1},
      {-2,  0,  2},
      {-1,  0,  1}
   };

   __constant__ float const filter_v2_y [3][3] = {
      { 1,  2,  1},
      { 0,  0,  0},
      {-1, -2, -1}
   };
}

__global__ void sobel_filter_kernel_v2 (int const w, int const h, pfc::byte_t * const dp_dst, pfc::byte_t const * const dp_src) {

// position of the thread in the image

   int const x = pfc::global_thread_idx_x ();
   int const y = pfc::global_thread_idx_y ();

   if (pfc::inside_rect (x, y, w, h)) {   // early exit if the thread is not in the image

//    load my eight neighbors and myself

      float n [3][3];   // = {}   // initialization costs time

      for (int j = -1; j <= 1 ; ++j) {
         for (int i = -1; i <= 1; ++i) {
            n[j + 1][i + 1] = pfc::inside_rect (x + i, y + j, w, h) ? dp_src[(y + j) * w + (x + i)] : 0;
         }
      }

//    compute the convolution

      float gx = 0;
      float gy = 0;

      for (int j = 0; j < 3; ++j) {
         for (int i = 0; i < 3; ++i) {
            gx = __fmaf_rd (filter_v2_x[j][i], n[j][i], gx);
            gy = __fmaf_rd (filter_v2_y[j][i], n[j][i], gy);
         }
      }

//    compute the gradient and store the result

      dp_dst[y * w + x] = pfc::min (255.0f, __fsqrt_rn (gx * gx + gy * gy));
   }
}

// -------------------------------------------------------------------------------------------------

void sobel_filter_kernel_v1_wrapper (pfc::byte_t * const dp_dst, pfc::byte_t const * const dp_src) {
   sobel_filter_kernel_v1
      <<<pfc::grid_size (pfc::config::block_size_sobel ()), pfc::config::block_size_sobel ()>>>
      (PFC_CONFIG.img_w, PFC_CONFIG.img_h, dp_dst, dp_src);

   PFC_CUDA_CHECK (hipGetLastError ());
}

void sobel_filter_kernel_v2_wrapper (pfc::byte_t * const dp_dst, pfc::byte_t const * const dp_src) {
   sobel_filter_kernel_v2
      <<<pfc::grid_size (pfc::config::block_size_sobel ()), pfc::config::block_size_sobel ()>>>
      (PFC_CONFIG.img_w, PFC_CONFIG.img_h, dp_dst, dp_src);

   PFC_CUDA_CHECK (hipGetLastError ());
}
