#include "hip/hip_runtime.h"
//       $Id: rgba_to_gray.cu 30314 2016-01-03 15:42:34Z p20068 $
//      $URL: https://svn01.fh-hagenberg.at/bin/cepheiden/vocational/teaching/ESD/SPS3/2015-WS/Ablauf/src/Filters/src/device/rgba_to_gray.cu $
// $Revision: 30314 $
//     $Date: 2016-01-03 16:42:34 +0100 (So., 03 Jän 2016) $
//   Creator: Ch. Angerer, NVIDIA, J. Demouth, NVIDIA, peter.kulczycki<AT>fh-hagenberg.at
//   $Author: p20068 $

#include "./rgba_to_gray.h"

// -------------------------------------------------------------------------------------------------

__global__ void rgba_to_gray_kernel_v1 (int const w, int const h, pfc::byte_t * const dp_dst, pfc::pixel_t const * const dp_src) {

// position of the thread in the image

   int const x = pfc::global_thread_idx_x ();
   int const y = pfc::global_thread_idx_y ();

   if (pfc::inside_rect (x, y, w, h)) {   // early exit if the thread is not in the image

//    load the pixel in RGBA format

      pfc::pixel_t const p = dp_src[y * w + x];

//    extract the three components in FP32

      float const r = p.x;
      float const g = p.y;
      float const b = p.z;

//    compute the gray value store the result

      dp_dst[y * w + x] = pfc::gray_value (r, g, b);   // make_uchar4 (gray, gray, gray, 0)
   }
}

// -------------------------------------------------------------------------------------------------

__global__ void rgba_to_gray_kernel_v2 (int const w, int const h, pfc::byte_t * const dp_dst, pfc::pixel_t const * const dp_src) {

// position of the thread in the image, compute two pixels at the same time

   int const x =     pfc::global_thread_idx_x ();
   int const y = 2 * pfc::global_thread_idx_y ();

// load the pixels in RGBA format

   pfc::pixel_t const p0 = pfc::inside_rect (x, y, w, h) ? dp_src[(y + 0) * w + x] : make_uchar4 (0, 0, 0, 0);
   pfc::pixel_t const p1 = pfc::inside_rect (x, y, w, h) ? dp_src[(y + 1) * w + x] : make_uchar4 (0, 0, 0, 0);

// extract the three components in FP32

   float const r0 = p0.x;
   float const g0 = p0.y;
   float const b0 = p0.z;

   float const r1 = p1.x;
   float const g1 = p1.y;
   float const b1 = p1.z;

// compute the gray values and store the results

   if (pfc::inside_rect (x, y + 0, w, h)) dp_dst[(y + 0) * w + x] = pfc::gray_value (r0, g0, b0);
   if (pfc::inside_rect (x, y + 1, w, h)) dp_dst[(y + 1) * w + x] = pfc::gray_value (r1, g1, b1);
}

// -------------------------------------------------------------------------------------------------

void rgba_to_gray_kernel_v1_wrapper (pfc::byte_t * const dp_dst, pfc::pixel_t const * const dp_src) {
   rgba_to_gray_kernel_v1
      <<<pfc::grid_size (pfc::config::block_size_gray ()), pfc::config::block_size_gray ()>>>
      (PFC_CONFIG.img_w, PFC_CONFIG.img_h, dp_dst, dp_src);

   PFC_CUDA_CHECK (hipGetLastError ());
}

void rgba_to_gray_kernel_v2_wrapper (pfc::byte_t * const dp_dst, pfc::pixel_t const * const dp_src) {
   rgba_to_gray_kernel_v2
      <<<pfc::grid_size (pfc::config::block_size_gray ()), pfc::config::block_size_gray ()>>>
      (PFC_CONFIG.img_w, PFC_CONFIG.img_h, dp_dst, dp_src);

   PFC_CUDA_CHECK (hipGetLastError ());
}
