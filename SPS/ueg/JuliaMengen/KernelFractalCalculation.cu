#include "hip/hip_runtime.h"
#include "KernelFractalCalculation.cuh"

__global__ void FractalCalculation(pfc::complex<double> const c, pfc::RGB_3_t * dp_data,
    size_t const dataSize, size_t const imageWidth, size_t const imageHeight,
    pfc::RGB_3_t * dp_colorTable, size_t const colorTableSize,
    pfc::complex<double> const lowerLeft, double const stepX, double const stepY,
    julia::JuliaPixelCalculation<int, double> const * dp_pixelCalc)
{
    julia::JuliaFractalCalculationCUDA<int, double>::CalcPixel(blockIdx, blockDim, threadIdx, c, dp_data,
        dataSize, imageWidth, imageHeight, dp_colorTable, colorTableSize, lowerLeft, stepX, stepY,
        dp_pixelCalc);
}

__global__ void FractalCalculation(pfc::complex<float> const c, pfc::RGB_3_t * dp_data,
    size_t const dataSize, size_t const imageWidth, size_t const imageHeight,
    pfc::RGB_3_t * dp_colorTable, size_t const colorTableSize,
    pfc::complex<float> const lowerLeft, float const stepX, float const stepY,
    julia::JuliaPixelCalculation<int, float> const * dp_pixelCalc)
{
    julia::JuliaFractalCalculationCUDA<int, float>::CalcPixel(blockIdx, blockDim, threadIdx, c, dp_data,
        dataSize, imageWidth, imageHeight, dp_colorTable, colorTableSize, lowerLeft, stepX, stepY,
        dp_pixelCalc);
}

__host__ void FractalCalculationWrapper(dim3 const blockDimension, size_t const threadPerBlock,
    pfc::complex<double> const c, pfc::RGB_3_t * dp_data, size_t const dataSize,
    size_t const imageWidth, size_t const imageHeight, pfc::RGB_3_t * dp_colorTable,
    size_t const colorTableSize, pfc::complex<double> const lowerLeft, double const stepX,
    double const stepY, julia::JuliaPixelCalculation<int, double> const * dp_pixelCalc)
{
    FractalCalculation << < blockDimension, threadPerBlock >> > (c, dp_data, dataSize, imageWidth,
        imageHeight, dp_colorTable, colorTableSize, lowerLeft, stepX, stepY, dp_pixelCalc);
}

__host__ void FractalCalculationWrapper(dim3 const blockDimension, size_t const threadPerBlock,
    pfc::complex<float> const c, pfc::RGB_3_t * dp_data, size_t const dataSize,
    size_t const imageWidth, size_t const imageHeight, pfc::RGB_3_t * dp_colorTable,
    size_t const colorTableSize, pfc::complex<float> const lowerLeft, float const stepX,
    float const stepY, julia::JuliaPixelCalculation<int, float> const * dp_pixelCalc)
{
    FractalCalculation << < blockDimension, threadPerBlock >> > (c, dp_data, dataSize, imageWidth,
        imageHeight, dp_colorTable, colorTableSize, lowerLeft, stepX, stepY, dp_pixelCalc);
}