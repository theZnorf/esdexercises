#include "hip/hip_runtime.h"
#include "KernelFractalCalculation.cuh"
#include "bitmap.h"
#include "cudautils.h"

//////////////////// Version 0 ///////////////////////////////
__global__ void FractalCalculationV0(pfc::complex<double> const c, pfc::RGB_3_t * dp_data,
    size_t const dataSize, size_t const imageWidth, size_t const imageHeight,
    pfc::RGB_3_t * dp_colorTable, size_t const colorTableSize,
    pfc::complex<double> const lowerLeft, double const stepX, double const stepY,
    julia::JuliaPixelCalculation<int, double> const * dp_pixelCalc)
{
    julia::JuliaFractalCalculationCUDA<int, double>::CalcPixelV0(blockIdx, blockDim, threadIdx, c, dp_data,
        dataSize, imageWidth, imageHeight, dp_colorTable, colorTableSize, lowerLeft, stepX, stepY,
        dp_pixelCalc);
}

__global__ void FractalCalculationV0(pfc::complex<float> const c, pfc::RGB_3_t * dp_data,
    size_t const dataSize, size_t const imageWidth, size_t const imageHeight,
    pfc::RGB_3_t * dp_colorTable, size_t const colorTableSize,
    pfc::complex<float> const lowerLeft, float const stepX, float const stepY,
    julia::JuliaPixelCalculation<int, float> const * dp_pixelCalc)
{
    julia::JuliaFractalCalculationCUDA<int, float>::CalcPixelV0(blockIdx, blockDim, threadIdx, c, dp_data,
        dataSize, imageWidth, imageHeight, dp_colorTable, colorTableSize, lowerLeft, stepX, stepY,
        dp_pixelCalc);
}


__host__ void FractalCalculationWrapperV0(dim3 const blockDimension, size_t const threadPerBlock,
    pfc::complex<double> const c, pfc::RGB_3_t * dp_data, size_t const dataSize,
    size_t const imageWidth, size_t const imageHeight, pfc::RGB_3_t * dp_colorTable,
    size_t const colorTableSize, pfc::complex<double> const lowerLeft, double const stepX,
    double const stepY, julia::JuliaPixelCalculation<int, double> const * dp_pixelCalc)
{
    FractalCalculationV0 <<< blockDimension, threadPerBlock >>> (c, dp_data, dataSize, imageWidth,
        imageHeight, dp_colorTable, colorTableSize, lowerLeft, stepX, stepY, dp_pixelCalc);
}

__host__ void FractalCalculationWrapperV0(dim3 const blockDimension, size_t const threadPerBlock,
    pfc::complex<float> const c, pfc::RGB_3_t * dp_data, size_t const dataSize,
    size_t const imageWidth, size_t const imageHeight, pfc::RGB_3_t * dp_colorTable,
    size_t const colorTableSize, pfc::complex<float> const lowerLeft, float const stepX,
    float const stepY, julia::JuliaPixelCalculation<int, float> const * dp_pixelCalc)
{
    FractalCalculationV0 <<< blockDimension, threadPerBlock >>> (c, dp_data, dataSize, imageWidth,
        imageHeight, dp_colorTable, colorTableSize, lowerLeft, stepX, stepY, dp_pixelCalc);
}


///////////////////////////// Version 1 /////////////////////////////////////////
__constant__ pfc::RGB_3_t constColorTableV1[512];


__global__ void FractalCalculationV1(pfc::complex<double> const c, pfc::RGB_3_t * dp_data,
    size_t const dataSize, size_t const imageWidth, size_t const imageHeight, size_t const colorTableSize,
    pfc::complex<double> const lowerLeft, double const stepX, double const stepY,
    julia::JuliaPixelCalculation<int, double> const * dp_pixelCalc)
{
    julia::JuliaFractalCalculationCUDA<int, double>::CalcPixelV0(blockIdx, blockDim, threadIdx, c, dp_data,
        dataSize, imageWidth, imageHeight, constColorTableV1, colorTableSize, lowerLeft, stepX, stepY,
        dp_pixelCalc);
}

__global__ void FractalCalculationV1(pfc::complex<float> const c, pfc::RGB_3_t * dp_data,
    size_t const dataSize, size_t const imageWidth, size_t const imageHeight, size_t const colorTableSize,
    pfc::complex<float> const lowerLeft, float const stepX, float const stepY,
    julia::JuliaPixelCalculation<int, float> const * dp_pixelCalc)
{
    julia::JuliaFractalCalculationCUDA<int, float>::CalcPixelV0(blockIdx, blockDim, threadIdx, c, dp_data,
        dataSize, imageWidth, imageHeight, constColorTableV1, colorTableSize, lowerLeft, stepX, stepY,
        dp_pixelCalc);
}

__host__ void FractalCalculationWrapperV1(dim3 const blockDimension, size_t const threadPerBlock,
    pfc::complex<double> const c, pfc::RGB_3_t * dp_data, size_t const dataSize,
    size_t const imageWidth, size_t const imageHeight, pfc::RGB_3_t * p_colorTable,
    size_t const colorTableSize, pfc::complex<double> const lowerLeft, double const stepX,
    double const stepY, julia::JuliaPixelCalculation<int, double> const * dp_pixelCalc)
{
    // copy color table
    cudautils::symbolmemcopy(constColorTableV1, p_colorTable, colorTableSize);

    FractalCalculationV1 <<< blockDimension, threadPerBlock >>> (c, dp_data, dataSize, imageWidth,
        imageHeight, colorTableSize, lowerLeft, stepX, stepY, dp_pixelCalc);
}

__host__ void FractalCalculationWrapperV1(dim3 const blockDimension, size_t const threadPerBlock,
    pfc::complex<float> const c, pfc::RGB_3_t * dp_data, size_t const dataSize,
    size_t const imageWidth, size_t const imageHeight, pfc::RGB_3_t * p_colorTable,
    size_t const colorTableSize, pfc::complex<float> const lowerLeft, float const stepX,
    float const stepY, julia::JuliaPixelCalculation<int, float> const * dp_pixelCalc)
{
    // copy color table
    cudautils::symbolmemcopy(constColorTableV1, p_colorTable, colorTableSize);

    FractalCalculationV1 <<< blockDimension, threadPerBlock >>> (c, dp_data, dataSize, imageWidth,
        imageHeight, colorTableSize, lowerLeft, stepX, stepY, dp_pixelCalc);
}


///////////////////////////// Version 2 /////////////////////////////////////////

__global__ void FractalCalculationV2(pfc::complex<double> const c, pfc::RGB_3_t * dp_data,
    size_t const dataSize, size_t const imageWidth, size_t const imageHeight, size_t const colorTableSize,
    pfc::complex<double> const lowerLeft, double const stepX, double const stepY,
    julia::JuliaPixelCalculation<int, double> const * dp_pixelCalc)
{
    julia::JuliaFractalCalculationCUDA<int, double>::CalcPixelV2(blockIdx, blockDim, threadIdx, c, dp_data,
        dataSize, imageWidth, imageHeight, constColorTableV1, colorTableSize, lowerLeft, stepX, stepY,
        dp_pixelCalc);
}

__global__ void FractalCalculationV2(pfc::complex<float> const c, pfc::RGB_3_t * dp_data,
    size_t const dataSize, size_t const imageWidth, size_t const imageHeight, size_t const colorTableSize,
    pfc::complex<float> const lowerLeft, float const stepX, float const stepY,
    julia::JuliaPixelCalculation<int, float> const * dp_pixelCalc)
{
    julia::JuliaFractalCalculationCUDA<int, float>::CalcPixelV2(blockIdx, blockDim, threadIdx, c, dp_data,
        dataSize, imageWidth, imageHeight, constColorTableV1, colorTableSize, lowerLeft, stepX, stepY,
        dp_pixelCalc);
}

__host__ void FractalCalculationWrapperV2(dim3 const blockDimension, size_t const threadPerBlock,
    pfc::complex<double> const c, pfc::RGB_3_t * dp_data, size_t const dataSize,
    size_t const imageWidth, size_t const imageHeight, pfc::RGB_3_t * p_colorTable,
    size_t const colorTableSize, pfc::complex<double> const lowerLeft, double const stepX,
    double const stepY, julia::JuliaPixelCalculation<int, double> const * dp_pixelCalc)
{
    // copy color table
    cudautils::symbolmemcopy(constColorTableV1, p_colorTable, colorTableSize);

    FractalCalculationV2 << < blockDimension, threadPerBlock >> > (c, dp_data, dataSize, imageWidth,
        imageHeight, colorTableSize, lowerLeft, stepX, stepY, dp_pixelCalc);
}

__host__ void FractalCalculationWrapperV2(dim3 const blockDimension, size_t const threadPerBlock,
    pfc::complex<float> const c, pfc::RGB_3_t * dp_data, size_t const dataSize,
    size_t const imageWidth, size_t const imageHeight, pfc::RGB_3_t * p_colorTable,
    size_t const colorTableSize, pfc::complex<float> const lowerLeft, float const stepX,
    float const stepY, julia::JuliaPixelCalculation<int, float> const * dp_pixelCalc)
{
    // copy color table
    cudautils::symbolmemcopy(constColorTableV1, p_colorTable, colorTableSize);

    FractalCalculationV2 << < blockDimension, threadPerBlock >> > (c, dp_data, dataSize, imageWidth,
        imageHeight, colorTableSize, lowerLeft, stepX, stepY, dp_pixelCalc);
}

///////////////////////////// Version 3 /////////////////////////////////////////

__global__ void FractalCalculationV3(pfc::complex<double> const c, pfc::RGB_3_t * dp_data,
    size_t const dataSize, size_t const imageWidth, size_t const imageHeight, size_t const colorTableSize,
    pfc::complex<double> const lowerLeft, double const stepX, double const stepY,
    julia::JuliaPixelCalculation<int, double> const * dp_pixelCalc, size_t const pixelPerThread)
{
    // calculate pixel
    julia::JuliaFractalCalculationCUDA<int, double>::CalcPixelV3(blockIdx, blockDim, threadIdx, c, dp_data,
        dataSize, imageWidth, imageHeight, constColorTableV1, colorTableSize, lowerLeft, stepX, stepY,
        dp_pixelCalc, pixelPerThread);
}

__global__ void FractalCalculationV3(pfc::complex<float> const c, pfc::RGB_3_t * dp_data,
    size_t const dataSize, size_t const imageWidth, size_t const imageHeight, size_t const colorTableSize,
    pfc::complex<float> const lowerLeft, float const stepX, float const stepY,
    julia::JuliaPixelCalculation<int, float> const * dp_pixelCalc, size_t const pixelPerThread)
{
    // calculate pixel
    julia::JuliaFractalCalculationCUDA<int, float>::CalcPixelV3(blockIdx, blockDim, threadIdx, c, dp_data,
        dataSize, imageWidth, imageHeight, constColorTableV1, colorTableSize, lowerLeft, stepX, stepY,
        dp_pixelCalc, pixelPerThread);
}

__host__ void FractalCalculationWrapperV3(dim3 const blockDimension, size_t const threadPerBlock,
    pfc::complex<double> const c, pfc::RGB_3_t * dp_data, size_t const dataSize,
    size_t const imageWidth, size_t const imageHeight, pfc::RGB_3_t * p_colorTable,
    size_t const colorTableSize, pfc::complex<double> const lowerLeft, double const stepX,
    double const stepY, julia::JuliaPixelCalculation<int, double> const * dp_pixelCalc,
    size_t const pixelPerThread)
{
    // copy color table
    cudautils::symbolmemcopy(constColorTableV1, p_colorTable, colorTableSize);

    FractalCalculationV3 <<< blockDimension, threadPerBlock >>> (c, dp_data, dataSize, imageWidth,
        imageHeight, colorTableSize, lowerLeft, stepX, stepY, dp_pixelCalc, pixelPerThread);
}

__host__ void FractalCalculationWrapperV3(dim3 const blockDimension, size_t const threadPerBlock,
    pfc::complex<float> const c, pfc::RGB_3_t * dp_data, size_t const dataSize,
    size_t const imageWidth, size_t const imageHeight, pfc::RGB_3_t * p_colorTable,
    size_t const colorTableSize, pfc::complex<float> const lowerLeft, float const stepX,
    float const stepY, julia::JuliaPixelCalculation<int, float> const * dp_pixelCalc,
    size_t const pixelPerThread)
{
    // copy color table
    cudautils::symbolmemcopy(constColorTableV1, p_colorTable, colorTableSize);

    FractalCalculationV3 <<< blockDimension, threadPerBlock >>> (c, dp_data, dataSize, imageWidth,
        imageHeight, colorTableSize, lowerLeft, stepX, stepY, dp_pixelCalc, pixelPerThread);
}

///////////////////////////// Version 4 /////////////////////////////////////////

__global__ void FractalCalculationV4(pfc::complex<double> const c, pfc::RGB_3_t * dp_data,
    size_t const dataSize, size_t const imageWidth, size_t const imageHeight, size_t const colorTableSize,
    pfc::complex<double> const lowerLeft, double const stepX, double const stepY,
    julia::JuliaPixelCalculation<int, double> const * dp_pixelCalc, size_t const pixelPerThread)
{
    // calculate pixel
    julia::JuliaFractalCalculationCUDA<int, double>::CalcPixelV4(blockIdx, blockDim, threadIdx, c, dp_data,
        dataSize, imageWidth, imageHeight, constColorTableV1, colorTableSize, lowerLeft, stepX, stepY,
        dp_pixelCalc, pixelPerThread);
}

__global__ void FractalCalculationV4(pfc::complex<float> const c, pfc::RGB_3_t * dp_data,
    size_t const dataSize, size_t const imageWidth, size_t const imageHeight, size_t const colorTableSize,
    pfc::complex<float> const lowerLeft, float const stepX, float const stepY,
    julia::JuliaPixelCalculation<int, float> const * dp_pixelCalc, size_t const pixelPerThread)
{
    // calculate pixel
    julia::JuliaFractalCalculationCUDA<int, float>::CalcPixelV4(blockIdx, blockDim, threadIdx, c, dp_data,
        dataSize, imageWidth, imageHeight, constColorTableV1, colorTableSize, lowerLeft, stepX, stepY,
        dp_pixelCalc, pixelPerThread);
}

__host__ void FractalCalculationWrapperV4(dim3 const blockDimension, dim3 const threadDimension,
    pfc::complex<double> const c, pfc::RGB_3_t * dp_data, size_t const dataSize,
    size_t const imageWidth, size_t const imageHeight, pfc::RGB_3_t * p_colorTable,
    size_t const colorTableSize, pfc::complex<double> const lowerLeft, double const stepX,
    double const stepY, julia::JuliaPixelCalculation<int, double> const * dp_pixelCalc,
    size_t const pixelPerThread)
{
    // copy color table
    cudautils::symbolmemcopy(constColorTableV1, p_colorTable, colorTableSize);

    FractalCalculationV4 <<< blockDimension, threadDimension >>> (c, dp_data, dataSize, imageWidth,
        imageHeight, colorTableSize, lowerLeft, stepX, stepY, dp_pixelCalc, pixelPerThread);
}

__host__ void FractalCalculationWrapperV4(dim3 const blockDimension, dim3 const threadDimension,
    pfc::complex<float> const c, pfc::RGB_3_t * dp_data, size_t const dataSize,
    size_t const imageWidth, size_t const imageHeight, pfc::RGB_3_t * p_colorTable,
    size_t const colorTableSize, pfc::complex<float> const lowerLeft, float const stepX,
    float const stepY, julia::JuliaPixelCalculation<int, float> const * dp_pixelCalc,
    size_t const pixelPerThread)
{
    // copy color table
    cudautils::symbolmemcopy(constColorTableV1, p_colorTable, colorTableSize);

    FractalCalculationV4 <<< blockDimension, threadDimension >>> (c, dp_data, dataSize, imageWidth,
        imageHeight, colorTableSize, lowerLeft, stepX, stepY, dp_pixelCalc, pixelPerThread);
}


///////////////////////////// Version 5 /////////////////////////////////////////

__global__ void FractalCalculationV5(pfc::complex<double> const c, pfc::RGB_3_t * dp_data,
    size_t const dataSize, size_t const imageWidth, size_t const imageHeight, size_t const colorTableSize,
    pfc::complex<double> const lowerLeft, double const stepX, double const stepY,
    julia::JuliaPixelCalculation<int, double> const * dp_pixelCalc)
{
    // calculate pixel
    julia::JuliaFractalCalculationCUDA<int, double>::CalcPixelV5(blockIdx, blockDim, threadIdx, c, dp_data,
        dataSize, imageWidth, imageHeight, constColorTableV1, colorTableSize, lowerLeft, stepX, stepY,
        dp_pixelCalc);
}

__global__ void FractalCalculationV5(pfc::complex<float> const c, pfc::RGB_3_t * dp_data,
    size_t const dataSize, size_t const imageWidth, size_t const imageHeight, size_t const colorTableSize,
    pfc::complex<float> const lowerLeft, float const stepX, float const stepY,
    julia::JuliaPixelCalculation<int, float> const * dp_pixelCalc)
{
    // calculate pixel
    julia::JuliaFractalCalculationCUDA<int, float>::CalcPixelV5(blockIdx, blockDim, threadIdx, c, dp_data,
        dataSize, imageWidth, imageHeight, constColorTableV1, colorTableSize, lowerLeft, stepX, stepY,
        dp_pixelCalc);
}

__host__ void FractalCalculationWrapperV5(dim3 const blockDimension, dim3 const threadDimension,
    pfc::complex<double> const c, pfc::RGB_3_t * dp_data, size_t const dataSize,
    size_t const imageWidth, size_t const imageHeight, pfc::RGB_3_t * p_colorTable,
    size_t const colorTableSize, pfc::complex<double> const lowerLeft, double const stepX,
    double const stepY, julia::JuliaPixelCalculation<int, double> const * dp_pixelCalc)
{
    // copy color table
    cudautils::symbolmemcopy(constColorTableV1, p_colorTable, colorTableSize);

    FractalCalculationV5 << < blockDimension, threadDimension >> > (c, dp_data, dataSize, imageWidth,
        imageHeight, colorTableSize, lowerLeft, stepX, stepY, dp_pixelCalc);
}

__host__ void FractalCalculationWrapperV5(dim3 const blockDimension, dim3 const threadDimension,
    pfc::complex<float> const c, pfc::RGB_3_t * dp_data, size_t const dataSize,
    size_t const imageWidth, size_t const imageHeight, pfc::RGB_3_t * p_colorTable,
    size_t const colorTableSize, pfc::complex<float> const lowerLeft, float const stepX,
    float const stepY, julia::JuliaPixelCalculation<int, float> const * dp_pixelCalc)
{
    // copy color table
    cudautils::symbolmemcopy(constColorTableV1, p_colorTable, colorTableSize);

    FractalCalculationV5 << < blockDimension, threadDimension >> > (c, dp_data, dataSize, imageWidth,
        imageHeight, colorTableSize, lowerLeft, stepX, stepY, dp_pixelCalc);
}